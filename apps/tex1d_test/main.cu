#include "hip/hip_runtime.h"
#include <iostream>
#include <>
#include "common/sanity_check.h"
#include "common/common_utils.h"

__global__ void testTex1DFetchKernel(
	hipTextureObject_t tex,
	const unsigned num_elems,
	surfelwarp::device::PtrSz<float> fromTexture
) {
	const auto idx = threadIdx.x + blockDim.x * blockIdx.x;
	if(idx < num_elems) {
		fromTexture[idx] = surfelwarp::fetch1DLinear<float>(tex, idx);
	}
}

int main()
{
	using namespace surfelwarp;

	//Prepare the test data on host
	const auto test_size = 1500;
	std::vector<float> h_vec;
	h_vec.resize(test_size);
	fillRandomVector(h_vec);

	//Upload it to device
	DeviceArray<float> d_vec;
	d_vec.upload(h_vec);

	//Create texture desc
	hipTextureDesc texture_desc;
	memset(&texture_desc, 0, sizeof(hipTextureDesc));
	texture_desc.normalizedCoords = 0;
	texture_desc.addressMode[0] = hipAddressModeBorder;
	texture_desc.addressMode[1] = hipAddressModeBorder;
	texture_desc.addressMode[2] = hipAddressModeBorder;
	texture_desc.filterMode = hipFilterModePoint;
	texture_desc.readMode = hipReadModeElementType;
	texture_desc.sRGB = 0;

	//Create resource desc
	hipResourceDesc resource_desc;
	memset(&resource_desc, 0, sizeof(hipResourceDesc));
	resource_desc.resType = hipResourceTypeLinear;
	resource_desc.res.linear.devPtr = d_vec.ptr();
	resource_desc.res.linear.sizeInBytes = d_vec.sizeBytes();
	resource_desc.res.linear.desc.f = hipChannelFormatKindFloat;
	resource_desc.res.linear.desc.x = 32;
	resource_desc.res.linear.desc.y = 0;
	resource_desc.res.linear.desc.z = 0;
	resource_desc.res.linear.desc.w = 0;

	//The texture object
	hipTextureObject_t d_texture;
	cudaSafeCall(hipCreateTextureObject(&d_texture, &resource_desc, &texture_desc, nullptr));

	//Access from kernel
	DeviceArray<float> fromTexture;
	fromTexture.create(test_size);
	dim3 blk(128);
	dim3 grid(divUp(test_size, blk.x));
	testTex1DFetchKernel<<<grid, blk>>>(d_texture, test_size, fromTexture);
	hipDeviceSynchronize();

	//Compare the difference
	std::vector<float> h_from_text;
	fromTexture.download(h_from_text);
	auto err = maxRelativeError(h_from_text, h_vec);
	std::cout << "The err of text fetching iter 1 " << err << std::endl;

	//Check again with updated device array
	fillRandomVector(h_vec);
	d_vec.upload(h_vec);
	testTex1DFetchKernel<<<grid, blk>>>(d_texture, test_size, fromTexture);
	hipDeviceSynchronize();
	
	//Download again
	fromTexture.download(h_from_text);
	err = maxRelativeError(h_from_text, h_vec);
	std::cout << "The err of text fetching iter 1 " << err << std::endl;
	return 0;
}