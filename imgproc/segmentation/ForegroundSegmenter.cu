#include "hip/hip_runtime.h"
#include "imgproc/segmentation/ForegroundSegmenter.h"
#include <>

namespace surfelwarp { namespace device {
	
	__global__ void upsampleFilterForegroundMaskKernel(
		hipTextureObject_t subsampled_mask, 
		unsigned upsample_rows, unsigned upsample_cols,
		unsigned sample_rate,
		const float sigma,
		hipSurfaceObject_t upsampled_mask, 
		hipSurfaceObject_t filter_mask
	) {
		const int x = threadIdx.x + blockDim.x * blockIdx.x;
		const int y = threadIdx.y + blockDim.y * blockIdx.y;
		if(x >= upsample_cols || y >= upsample_rows) return;

		//A window search
		const int halfsize = __float2uint_ru(sigma) * 2;
		float total_weight = 0.0f;
		float total_value = 0.0f;
		for(int neighbor_y = y - halfsize; neighbor_y <= y + halfsize; neighbor_y++) {
			for(int neighbor_x = x - halfsize; neighbor_x <= x + halfsize; neighbor_x++) {
				//Retrieve the mask value at neigbour
				const auto subsampled_neighbor_x = neighbor_x / sample_rate;
				const auto subsampled_neighbor_y = neighbor_y / sample_rate;
				const unsigned char neighbor_foreground = tex2D<unsigned char>(subsampled_mask, subsampled_neighbor_x, subsampled_neighbor_y);

				//Compute the gaussian weight
				const float diff_x_square = (neighbor_x - x) * (neighbor_x - x);
				const float diff_y_square = (neighbor_y - y) * (neighbor_y - y);
				const float weight = __expf(0.5f * (diff_x_square + diff_y_square) / (sigma * sigma));

				//Accumlate it
				if(neighbor_x >= 0 && neighbor_x < upsample_cols && neighbor_y >= 0 && neighbor_y < upsample_rows)
				{
					total_weight += weight;
					total_value += weight * float(1 - neighbor_foreground);
				}
			}
		}

		
		//Compute the value locally
		const auto subsampled_x = x / sample_rate;
		const auto subsampled_y = y / sample_rate;
		const unsigned char foreground_indicator = tex2D<unsigned char>(subsampled_mask, subsampled_x, subsampled_y);
		float filter_value = 0.0;
		if(foreground_indicator == 0) {
			filter_value = total_value / (total_weight + 1e-3f);
		}
		

		//Write to the surface
		surf2Dwrite(foreground_indicator, upsampled_mask, x * sizeof(unsigned char), y);
		surf2Dwrite(filter_value, filter_mask, x * sizeof(float), y);
	}

	__global__ void filterForegroundMaskKernel(
		hipTextureObject_t foreground_mask, 
		unsigned mask_rows, unsigned mask_cols,
		const float sigma,
		hipSurfaceObject_t filter_mask
	) {
		const int x = threadIdx.x + blockDim.x * blockIdx.x;
		const int y = threadIdx.y + blockDim.y * blockIdx.y;
		if(x >= mask_cols || y >= mask_rows) return;

		//A window search
		const int halfsize = __float2uint_ru(sigma) * 2;
		float total_weight = 0.0f;
		float total_value = 0.0f;
		for(int neighbor_y = y - halfsize; neighbor_y <= y + halfsize; neighbor_y++) {
			for(int neighbor_x = x - halfsize; neighbor_x <= x + halfsize; neighbor_x++) {
				//Retrieve the mask value at neigbour
				const unsigned char neighbor_foreground = tex2D<unsigned char>(foreground_mask, neighbor_x, neighbor_y);

				//Compute the gaussian weight
				const float diff_x_square = (neighbor_x - x) * (neighbor_x - x);
				const float diff_y_square = (neighbor_y - y) * (neighbor_y - y);
				const float weight = __expf(0.5f * (diff_x_square + diff_y_square) / (sigma * sigma));

				//Accumlate it
				if(neighbor_x >= 0 && neighbor_x < mask_cols && neighbor_y >= 0 && neighbor_y < mask_rows)
				{
					total_weight += weight;
					total_value += weight * float(1 - neighbor_foreground);
				}
			}
		}

		
		//Compute the value locally
		const unsigned char foreground_indicator = tex2D<unsigned char>(foreground_mask, x, y);
		float filter_value = 0.0;
		if(foreground_indicator == 0) {
			filter_value = total_value / (total_weight + 1e-3f);
		}
		

		//Write to the surface
		surf2Dwrite(filter_value, filter_mask, x * sizeof(float), y);
	}

} // device
} // surfelwarp

void surfelwarp::ForegroundSegmenter::UpsampleFilterForegroundMask(
	hipTextureObject_t subsampled_mask,
	unsigned subsampled_rows, unsigned subsampled_cols,
	unsigned subsample_rate,
	float sigma,
	hipSurfaceObject_t upsampled_mask, 
	hipSurfaceObject_t filter_mask,
	hipStream_t stream
) {
	//Compute the size
	const auto upsampled_rows = subsampled_rows * subsample_rate;
	const auto upsampled_cols = subsampled_cols * subsample_rate;
	dim3 blk(16, 16);
	dim3 grid(divUp(upsampled_cols, blk.x), divUp(upsampled_rows, blk.y));

	//Invoke the kernel
	device::upsampleFilterForegroundMaskKernel<<<grid, blk, 0, stream>>>(
		subsampled_mask, 
		upsampled_rows, upsampled_cols, 
		subsample_rate, 
		sigma, 
		upsampled_mask, 
		filter_mask
	);

	//Sync and check error
#if defined(CUDA_DEBUG_SYNC_CHECK)
	cudaSafeCall(hipStreamSynchronize(stream));
	cudaSafeCall(hipGetLastError());
#endif
}


void surfelwarp::ForegroundSegmenter::FilterForegroundMask(
	hipTextureObject_t foreground_mask,
	unsigned mask_rows, unsigned mask_cols, 
	float sigma, 
	hipSurfaceObject_t filter_mask,
	hipStream_t stream
) {
	dim3 blk(16, 16);
	dim3 grid(divUp(mask_cols, blk.x), divUp(mask_rows, blk.y));
	device::filterForegroundMaskKernel<<<grid, blk, 0, stream>>>(
		foreground_mask, 
		mask_rows, mask_cols, 
		sigma, 
		filter_mask
	);

	//Sync and check error
#if defined(CUDA_DEBUG_SYNC_CHECK)
	cudaSafeCall(hipStreamSynchronize(stream));
	cudaSafeCall(hipGetLastError());
#endif
}