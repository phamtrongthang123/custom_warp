#include "common/common_texture_utils.h"

hipTextureObject_t surfelwarp::create1DLinearTexture(const DeviceArray<float> &array) {
	hipTextureDesc texture_desc;
	memset(&texture_desc, 0, sizeof(hipTextureDesc));
	texture_desc.normalizedCoords = 0;
	texture_desc.addressMode[0] = hipAddressModeBorder; //Return 0 outside the boundary
	texture_desc.addressMode[1] = hipAddressModeBorder;
	texture_desc.addressMode[2] = hipAddressModeBorder;
	texture_desc.filterMode = hipFilterModePoint;
	texture_desc.readMode = hipReadModeElementType;
	texture_desc.sRGB = 0;

	//Create resource desc
	hipResourceDesc resource_desc;
	memset(&resource_desc, 0, sizeof(hipResourceDesc));
	resource_desc.resType = hipResourceTypeLinear;
	resource_desc.res.linear.devPtr = (void*)array.ptr();
	resource_desc.res.linear.sizeInBytes = array.sizeBytes();
	resource_desc.res.linear.desc.f = hipChannelFormatKindFloat;
	resource_desc.res.linear.desc.x = 32;
	resource_desc.res.linear.desc.y = 0;
	resource_desc.res.linear.desc.z = 0;
	resource_desc.res.linear.desc.w = 0;

	//Allocate the texture
	hipTextureObject_t d_texture;
	cudaSafeCall(hipCreateTextureObject(&d_texture, &resource_desc, &texture_desc, nullptr));
	return d_texture;
}

hipTextureObject_t surfelwarp::create1DLinearTexture(const DeviceBufferArray<float>& array) {
	DeviceArray<float> pcl_array((float*)array.Ptr(), array.Capacity());
	return create1DLinearTexture(pcl_array);
}

void surfelwarp::createDefault2DTextureDesc(hipTextureDesc &desc) {
	memset(&desc, 0, sizeof(desc));
	desc.addressMode[0] = hipAddressModeBorder; //Return 0 outside the boundary
	desc.addressMode[1] = hipAddressModeBorder;
	desc.addressMode[2] = hipAddressModeBorder;
	desc.filterMode = hipFilterModePoint;
	desc.readMode = hipReadModeElementType;
	desc.normalizedCoords = 0;
}


void surfelwarp::createDepthTexture(
	const unsigned img_rows,
	const unsigned img_cols,
	hipTextureObject_t &texture,
	hipArray_t &d_array
) {
	//The texture description
	hipTextureDesc depth_texture_desc;
	createDefault2DTextureDesc(depth_texture_desc);

	//Create channel descriptions
	hipChannelFormatDesc depth_channel_desc = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindUnsigned);

	//Allocate the cuda array
	cudaSafeCall(hipMallocArray(&d_array, &depth_channel_desc, img_cols, img_rows));

	//Create the resource desc
	hipResourceDesc resource_desc;
	memset(&resource_desc, 0, sizeof(hipResourceDesc));
	resource_desc.resType = hipResourceTypeArray;
	resource_desc.res.array.array = d_array;

	//Allocate the texture
	cudaSafeCall(hipCreateTextureObject(&texture, &resource_desc, &depth_texture_desc, 0));
}



void surfelwarp::createDepthTextureSurface(
	const unsigned img_rows,
	const unsigned img_cols,
	hipTextureObject_t &texture,
	hipSurfaceObject_t &surface,
	hipArray_t &d_array
) {
	//The texture description
	hipTextureDesc depth_texture_desc;
	createDefault2DTextureDesc(depth_texture_desc);

	//Create channel descriptions
	hipChannelFormatDesc depth_channel_desc = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindUnsigned);

	//Allocate the cuda array
	cudaSafeCall(hipMallocArray(&d_array, &depth_channel_desc, img_cols, img_rows));

	//Create the resource desc
	hipResourceDesc resource_desc;
	memset(&resource_desc, 0, sizeof(hipResourceDesc));
	resource_desc.resType = hipResourceTypeArray;
	resource_desc.res.array.array = d_array;

	//Allocate the texture
	cudaSafeCall(hipCreateTextureObject(&texture, &resource_desc, &depth_texture_desc, 0));
	cudaSafeCall(hipCreateSurfaceObject(&surface, &resource_desc));
}


void surfelwarp::createDepthTextureSurface(const unsigned img_rows, const unsigned img_cols, CudaTextureSurface & collect) {
	createDepthTextureSurface(
		img_rows, img_cols,
		collect.texture, collect.surface, collect.d_array);
}



void surfelwarp::createFloat4TextureSurface(
	const unsigned rows, const unsigned cols,
	hipTextureObject_t &texture,
	hipSurfaceObject_t &surface,
	hipArray_t &d_array
) {
	//The texture description
	hipTextureDesc float4_texture_desc;
	createDefault2DTextureDesc(float4_texture_desc);

	//Create channel descriptions
	hipChannelFormatDesc float4_channel_desc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);

	//Allocate the cuda array
	cudaSafeCall(hipMallocArray(&d_array, &float4_channel_desc, cols, rows));

	//Create the resource desc
	hipResourceDesc resource_desc;
	memset(&resource_desc, 0, sizeof(hipResourceDesc));
	resource_desc.resType = hipResourceTypeArray;
	resource_desc.res.array.array = d_array;

	//Allocate the texture
	cudaSafeCall(hipCreateTextureObject(&texture, &resource_desc, &float4_texture_desc, 0));
	cudaSafeCall(hipCreateSurfaceObject(&surface, &resource_desc));
}


void surfelwarp::createFloat4TextureSurface(const unsigned rows, const unsigned cols, CudaTextureSurface & texture_collect)
{
	createFloat4TextureSurface(
		rows, cols,
		texture_collect.texture,
		texture_collect.surface,
		texture_collect.d_array
	);
}


void surfelwarp::createFloat1TextureSurface(
	const unsigned rows, const unsigned cols, 
	hipTextureObject_t & texture, 
	hipSurfaceObject_t & surface, 
	hipArray_t & d_array
) {
	//The texture description
	hipTextureDesc float1_texture_desc;
	createDefault2DTextureDesc(float1_texture_desc);

	//Create channel descriptions
	hipChannelFormatDesc float1_channel_desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	//Allocate the cuda array
	cudaSafeCall(hipMallocArray(&d_array, &float1_channel_desc, cols, rows));

	//Create the resource desc
	hipResourceDesc resource_desc;
	memset(&resource_desc, 0, sizeof(hipResourceDesc));
	resource_desc.resType = hipResourceTypeArray;
	resource_desc.res.array.array = d_array;

	//Allocate the texture
	cudaSafeCall(hipCreateTextureObject(&texture, &resource_desc, &float1_texture_desc, 0));
	cudaSafeCall(hipCreateSurfaceObject(&surface, &resource_desc));
}


void surfelwarp::createFloat1TextureSurface(
	const unsigned rows, const unsigned cols, 
	CudaTextureSurface & texture_collect
) {
	createFloat1TextureSurface(
		rows, cols, 
		texture_collect.texture, 
		texture_collect.surface, 
		texture_collect.d_array
	);
}


void surfelwarp::createFloat2TextureSurface(
	const unsigned rows, const unsigned cols, 
	hipTextureObject_t & texture, 
	hipSurfaceObject_t & surface, 
	hipArray_t & d_array
) {
	//The texture description
	hipTextureDesc float2_texture_desc;
	createDefault2DTextureDesc(float2_texture_desc);

	//Create channel descriptions
	hipChannelFormatDesc float2_channel_desc = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindFloat);

	//Allocate the cuda array
	cudaSafeCall(hipMallocArray(&d_array, &float2_channel_desc, cols, rows));

	//Create the resource desc
	hipResourceDesc resource_desc;
	memset(&resource_desc, 0, sizeof(hipResourceDesc));
	resource_desc.resType = hipResourceTypeArray;
	resource_desc.res.array.array = d_array;

	//Allocate the texture
	cudaSafeCall(hipCreateTextureObject(&texture, &resource_desc, &float2_texture_desc, 0));
	cudaSafeCall(hipCreateSurfaceObject(&surface, &resource_desc));
}

void surfelwarp::createFloat2TextureSurface(
	const unsigned rows, const unsigned cols, 
	CudaTextureSurface & texture_collect
) {
	createFloat2TextureSurface(
		rows, cols, 
		texture_collect.texture, 
		texture_collect.surface, 
		texture_collect.d_array
	);
}


void surfelwarp::createUChar1TextureSurface(
	const unsigned rows, const unsigned cols, 
	hipTextureObject_t & texture, 
	hipSurfaceObject_t & surface,
	hipArray_t & d_array
) {
	//The texture description
	hipTextureDesc uchar1_texture_desc;
	createDefault2DTextureDesc(uchar1_texture_desc);

	//Create channel descriptions
	hipChannelFormatDesc uchar1_channel_desc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);

	//Allocate the cuda array
	cudaSafeCall(hipMallocArray(&d_array, &uchar1_channel_desc, cols, rows));

	//Create the resource desc
	hipResourceDesc resource_desc;
	memset(&resource_desc, 0, sizeof(hipResourceDesc));
	resource_desc.resType = hipResourceTypeArray;
	resource_desc.res.array.array = d_array;

	//Allocate the texture
	cudaSafeCall(hipCreateTextureObject(&texture, &resource_desc, &uchar1_texture_desc, 0));
	cudaSafeCall(hipCreateSurfaceObject(&surface, &resource_desc));
}

void surfelwarp::createUChar1TextureSurface(
	const unsigned rows, const unsigned cols, 
	CudaTextureSurface & texture_collect
) {
	createUChar1TextureSurface(
		rows, cols, 
		texture_collect.texture, 
		texture_collect.surface, 
		texture_collect.d_array
	);
}



void surfelwarp::query2DTextureExtent(hipTextureObject_t texture, unsigned &width, unsigned &height) {
	hipResourceDesc texture_res;
	cudaSafeCall(hipGetTextureObjectResourceDesc(&texture_res, texture));
	hipArray_t cu_array = texture_res.res.array.array;
	hipChannelFormatDesc channel_desc;
	hipExtent extent;
	unsigned int flag;
	cudaSafeCall(hipArrayGetInfo(&channel_desc, &extent, &flag, cu_array));

	width = extent.width;
	height = extent.height;
}

void surfelwarp::releaseTextureCollect(CudaTextureSurface & texture_collect)
{
	cudaSafeCall(hipDestroyTextureObject(texture_collect.texture));
	cudaSafeCall(hipDestroySurfaceObject(texture_collect.surface));
	cudaSafeCall(hipFreeArray(texture_collect.d_array));
}

