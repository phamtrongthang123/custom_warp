#include "hip/hip_runtime.h"
#include "common/device_intrinsics.h"
#include "pcg_solver/solver_configs.h"
#include "core/warp_solver/term_offset_types.h"
#include "core/warp_solver/solver_encode.h"
#include "core/warp_solver/NodePair2TermsIndex.h"
#include <>
#include <math_functions.h>

namespace surfelwarp { namespace device {

	//Kernel for computing of the row offset in node_pair array
	__global__ void computeRowOffsetKernel(
		const DeviceArrayView<unsigned> compacted_Iij_key,
		DeviceArraySlice<unsigned> rowoffset_array
	) {
		const auto idx = threadIdx.x + blockIdx.x*blockDim.x;
		if (idx >= compacted_Iij_key.Size()) return;
		if (idx == 0) {
			rowoffset_array[0] = 0;
			rowoffset_array[rowoffset_array.Size() - 1] = compacted_Iij_key.Size();
		}
		else {
			const auto key_prev = compacted_Iij_key[idx - 1];
			const auto key_this = compacted_Iij_key[idx];
			const auto row_prev = encoded_row(key_prev);
			const auto row_this = encoded_row(key_this);
			if (row_this != row_prev) {
				rowoffset_array[row_this] = idx;
			}
		}
	}

	//Kernel for computing the length of each row 
	// (both diag and non-diagonal terms)
	__global__ void computeRowBlockLengthKernel(
		const unsigned* rowoffset_array,
		DeviceArraySlice<unsigned> blk_rowlength
	) {
		const auto idx = threadIdx.x + blockDim.x*blockIdx.x;
		if (idx >= blk_rowlength.Size()) return;
		//Note that the diagonal term is included
		blk_rowlength[idx] = 1 + rowoffset_array[idx + 1] - rowoffset_array[idx];
	}

	__global__ void computeBinLengthKernel(
		const DeviceArrayView<unsigned> rowblk_length,
		DeviceArraySlice<unsigned> valid_bin_length,
		unsigned* valid_nonzeros_rowscan
	) {
		__shared__ unsigned partial_sum[32];

		//The idx is in [0, 1024)
		const auto idx = threadIdx.x + blockDim.x*blockIdx.x;
		const auto warp_idx = idx >> 5;
		const auto lane_idx = idx & 31;
		
		unsigned bin_length = 0;
		if (idx < valid_bin_length.Size()) {
			//32 * idx is the real-matrix begin row
			//so does 32 * idx + 31 is the ending row
			//For a matrix row, its corresponding blk-row is
			//matrix_row / 6
			const unsigned blkrow_begin = bin_size * idx / 6;
			unsigned blkrow_end = (bin_size * idx + bin_size - 1) / 6;
			blkrow_end = umin(blkrow_end, rowblk_length.Size() - 1);
			unsigned max_length = 0;
			for (unsigned blkrow_idx = blkrow_begin; blkrow_idx <= blkrow_end; blkrow_idx++) {
				max_length = umax(max_length, rowblk_length[blkrow_idx]);
			}
			
			//From block length to actual element length
			bin_length = 6 * max_length;
			valid_bin_length[idx] = bin_length;
		}

		bin_length = warp_scan(bin_length);
		if (lane_idx == 31) {
			partial_sum[warp_idx] = bin_length;
		}
		__syncthreads();

		if (warp_idx == 0) {
			const unsigned partial_scan = partial_sum[lane_idx];
			partial_sum[lane_idx] = warp_scan(partial_scan);
		}
		__syncthreads();

		if (idx < valid_bin_length.Size()) {
			const unsigned offset = (warp_idx == 0 ? 0 : partial_sum[warp_idx - 1]);
			valid_nonzeros_rowscan[idx + 1] = 32 * (bin_length + offset);
		}
		
		//The first elements
		if (idx == 0) valid_nonzeros_rowscan[0] = 0;
	}


	__global__ void computeBinBlockedCSRRowPtrKernel(
		const unsigned* valid_nonzeros_rowscan,
		DeviceArraySlice<int> csr_rowptr
	) {
		const auto idx = threadIdx.x + blockDim.x * blockIdx.x;
		if (idx >= csr_rowptr.Size()) return;
		const int bin_row_idx = idx / 32;
		const int bin_row_offset = idx % 32;
		csr_rowptr[idx] = bin_row_offset + valid_nonzeros_rowscan[bin_row_idx];
	}


	//The column index for bin-block csr format
	__global__ void computeBinBlockedCSRColPtrKernel(
		const unsigned matrix_size,
		const int* csr_rowptr,
		const unsigned* compacted_nodepair,
		const unsigned* blkrow_offset,
		int* csr_colptr
	) {
		const auto row_idx = threadIdx.x + blockDim.x * blockIdx.x;
		if(row_idx >= matrix_size) return;

		//From now, the query on rowptr should be safe
		const auto blkrow_idx = row_idx / 6;
		const auto data_offset = csr_rowptr[row_idx];
		const auto lane_idx = threadIdx.x & 31;

		//For the diagonal terms
		auto column_idx_offset = (data_offset - lane_idx) / 6 + lane_idx;
		csr_colptr[column_idx_offset] = 6 * blkrow_idx;
		column_idx_offset += bin_size;

		//For the non-diagonal terms
		auto Iij_begin = blkrow_offset[blkrow_idx];
		const auto Iij_end = blkrow_offset[blkrow_idx + 1];

		for (; Iij_begin < Iij_end; Iij_begin++, column_idx_offset += bin_size) {
			const auto Iij_key = compacted_nodepair[Iij_begin];
			const auto blkcol_idx = encoded_col(Iij_key);
			csr_colptr[column_idx_offset] = 6 * blkcol_idx;
		}
	}

} // namespace device
} // namespace surfelwarp


void surfelwarp::NodePair2TermsIndex::computeBlockRowLength(hipStream_t stream) {
	m_blkrow_offset_array.ResizeArrayOrException(m_num_nodes + 1);
	
	//Prepare the input
	DeviceArrayView<unsigned> compacted_nodepair(m_symmetric_kv_sorter.valid_sorted_key);
	dim3 offset_blk(128);
	dim3 offset_grid(divUp(compacted_nodepair.Size(), offset_blk.x));
	device::computeRowOffsetKernel<<<offset_grid, offset_blk, 0, stream>>>(
		compacted_nodepair,
		m_blkrow_offset_array.ArraySlice()
	);
	
	//Sync and check error
#if defined(CUDA_DEBUG_SYNC_CHECK)
	cudaSafeCall(hipStreamSynchronize(stream));
	cudaSafeCall(hipGetLastError());
#endif
	
	//Compute the row length
	m_blkrow_length_array.ResizeArrayOrException(m_num_nodes);
	dim3 length_blk(64);
	dim3 length_grid(divUp(m_num_nodes, length_blk.x));
	device::computeRowBlockLengthKernel<<<length_grid, length_blk, 0, stream>>>(
		m_blkrow_offset_array.ArrayView(),
		m_blkrow_length_array.ArraySlice()
	);
	
	//Sync and check error
#if defined(CUDA_DEBUG_SYNC_CHECK)
	cudaSafeCall(hipStreamSynchronize(stream));
	cudaSafeCall(hipGetLastError());
#endif
	
	//Debug sanity check
	//blockRowOffsetSanityCheck();
	//blockRowLengthSanityCheck();
}


void surfelwarp::NodePair2TermsIndex::computeBinLength(hipStream_t stream) {
	//Correct the size of the matrix
	const auto matrix_size = m_num_nodes * 6;
	const auto num_bins = divUp(matrix_size, bin_size);
	
	m_binlength_array.ResizeArrayOrException(num_bins);
	m_binnonzeros_prefixsum.ResizeArrayOrException(num_bins + 1);
	device::computeBinLengthKernel<<<1, 1024, 0, stream>>>(
		m_blkrow_length_array.ArrayView(),
		m_binlength_array.ArraySlice(),
		m_binnonzeros_prefixsum.ArraySlice()
	);
	
	//Sync and check error
#if defined(CUDA_DEBUG_SYNC_CHECK)
	cudaSafeCall(hipStreamSynchronize(stream));
	cudaSafeCall(hipGetLastError());
#endif
	
	//The sanity check method
	//binLengthNonzerosSanityCheck();
}


void surfelwarp::NodePair2TermsIndex::computeBinBlockCSRRowPtr(hipStream_t stream) {
	//Compute the row pointer in bin-blocked csr format
	m_binblocked_csr_rowptr.ResizeArrayOrException(32 * m_binnonzeros_prefixsum.ArraySize());
	dim3 rowptr_blk(128);
	dim3 rowptr_grid(divUp(m_binblocked_csr_rowptr.ArraySize(), rowptr_blk.x));
	device::computeBinBlockedCSRRowPtrKernel<<<rowptr_grid, rowptr_blk, 0, stream>>>(
		m_binnonzeros_prefixsum.Ptr(),
		m_binblocked_csr_rowptr.ArraySlice()
	);
	
	//Sync and check error
#if defined(CUDA_DEBUG_SYNC_CHECK)
	cudaSafeCall(hipStreamSynchronize(stream));
	cudaSafeCall(hipGetLastError());
#endif
	
	//Sanity check method
	//binBlockCSRRowPtrSanityCheck();
}

void surfelwarp::NodePair2TermsIndex::nullifyBinBlockCSRColumePtr(hipStream_t stream) {
	//Compute the size to nullify
	const auto total_blk_size = m_symmetric_kv_sorter.valid_sorted_key.size() + m_num_nodes;
	const auto nullify_size = std::min(7 * total_blk_size, m_binblocked_csr_colptr.BufferSize());
	
	//Do it
	cudaSafeCall(hipMemsetAsync(
		m_binblocked_csr_colptr.Ptr(),
		0xFF,
		sizeof(int) * m_binblocked_csr_colptr.BufferSize(),
		stream
	));
}


void surfelwarp::NodePair2TermsIndex::computeBinBlockCSRColumnPtr(hipStream_t stream) {
	//The compacted full nodepair array
	DeviceArrayView<unsigned> compacted_nodepair(m_symmetric_kv_sorter.valid_sorted_key);
	const auto matrix_size = 6 * m_num_nodes;
	
	//Do not need to query the size of colptr?
	dim3 colptr_blk(128);
	dim3 colptr_grid(divUp(32 * m_binlength_array.ArraySize(), colptr_blk.x));
	device::computeBinBlockedCSRColPtrKernel<<<colptr_grid, colptr_blk, 0, stream>>>(
		matrix_size,
		m_binblocked_csr_rowptr.Ptr(),
		compacted_nodepair.RawPtr(),
		m_blkrow_offset_array.Ptr(),
		m_binblocked_csr_colptr.Ptr()
	);
	
	//Sync and check error
#if defined(CUDA_DEBUG_SYNC_CHECK)
	cudaSafeCall(hipStreamSynchronize(stream));
	cudaSafeCall(hipGetLastError());
#endif
	
	//Debug method
	//binBlockCSRColumnPtrSanityCheck();
}