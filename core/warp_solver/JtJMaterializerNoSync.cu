#include "hip/hip_runtime.h"
#include "common/ConfigParser.h"
#include "common/Constants.h"
#include "common/device_intrinsics.h"
#include "core/warp_solver/solver_encode.h"
#include "core/warp_solver/solver_constants.h"
#include "core/warp_solver/geometry_icp_jacobian.cuh"
#include "core/warp_solver/JtJMaterializer.h"
#include "core/warp_solver/jtj_block_jacobian.cuh"
#include <>

namespace surfelwarp { namespace device {

	enum {
		jtj_blk_size = 36,
		warp_size = 32,
		num_warps = 4,
		thread_blk_size = num_warps * warp_size,
	};
	
	__device__ __forceinline__ void computeScalarJtJBlock(
		const float jacobian[6],
		float jtj_blk[jtj_blk_size],
		const float weight_square = 1.0f
	) {
#pragma unroll
		for (int jac_row = 0; jac_row < 6; jac_row++) {
			jtj_blk[6 * jac_row + 0] = weight_square * jacobian[0] * jacobian[jac_row];
			jtj_blk[6 * jac_row + 1] = weight_square * jacobian[1] * jacobian[jac_row];
			jtj_blk[6 * jac_row + 2] = weight_square * jacobian[2] * jacobian[jac_row];
			jtj_blk[6 * jac_row + 3] = weight_square * jacobian[3] * jacobian[jac_row];
			jtj_blk[6 * jac_row + 4] = weight_square * jacobian[4] * jacobian[jac_row];
			jtj_blk[6 * jac_row + 5] = weight_square * jacobian[5] * jacobian[jac_row];
		}
	}

	__device__ __forceinline__ void computeSmoothJtJBlock(
		const NodeGraphSmoothTerm2Jacobian& term2jacobian,
		unsigned typed_term,
		unsigned encoded_pair,
		float jtj_blk[jtj_blk_size],
		const float weight_square = 1.0f
	) {
		//Check the validity of this term
		const auto validity = term2jacobian.validity_indicator[typed_term];
		if(validity == 0) {
#pragma unroll
			for (auto i = 0; i < jtj_blk_size; i++) {
				jtj_blk[i] = 0.0f;
			}
			return;
		}

		const ushort2 node_ij = term2jacobian.node_graph[typed_term];
		unsigned node_i, node_j;
		decode_nodepair(encoded_pair, node_i, node_j);

		//Explicit compute jacobian
		const float3 r = term2jacobian.Ti_xj[typed_term];
		const float3 s = term2jacobian.Tj_xj[typed_term];
		TwistGradientOfScalarCost twist_gradient_i, twist_gradient_j;
		
		//The order of two terms
		const float* jacobian_encoded_i;
		const float* jacobian_encoded_j;
		if(node_i == node_ij.x) {
			jacobian_encoded_i = (const float*)(&twist_gradient_i);
			jacobian_encoded_j = (const float*)(&twist_gradient_j);
		} else {
			jacobian_encoded_i = (const float*)(&twist_gradient_j);
			jacobian_encoded_j = (const float*)(&twist_gradient_i);
		}

		//The first iteration assign
		{
			twist_gradient_i.rotation = make_float3(0.0f, r.z, -r.y);
			twist_gradient_i.translation = make_float3(1.0f, 0.0f, 0.0f);
			twist_gradient_j.rotation = make_float3(0.0f, -s.z, s.y);
			twist_gradient_j.translation = make_float3(-1.0f,  0.0f,  0.0f);
#pragma unroll
			for (int jac_row = 0; jac_row < 6; jac_row++) {
				jtj_blk[6 * jac_row + 0] = weight_square * jacobian_encoded_i[0] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 1] = weight_square * jacobian_encoded_i[1] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 2] = weight_square * jacobian_encoded_i[2] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 3] = weight_square * jacobian_encoded_i[3] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 4] = weight_square * jacobian_encoded_i[4] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 5] = weight_square * jacobian_encoded_i[5] * jacobian_encoded_j[jac_row];
			}
		}

		//The next two iterations, plus
		{
			twist_gradient_i.rotation = make_float3(-r.z, 0.0f, r.x);
			twist_gradient_i.translation = make_float3(0.0f, 1.0f, 0.0f);
			twist_gradient_j.rotation = make_float3(s.z, 0.0f, -s.x);
			twist_gradient_j.translation = make_float3( 0.0f, -1.0f,  0.0f);
#pragma unroll
			for (int jac_row = 0; jac_row < 6; jac_row++) {
				jtj_blk[6 * jac_row + 0] += weight_square * jacobian_encoded_i[0] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 1] += weight_square * jacobian_encoded_i[1] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 2] += weight_square * jacobian_encoded_i[2] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 3] += weight_square * jacobian_encoded_i[3] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 4] += weight_square * jacobian_encoded_i[4] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 5] += weight_square * jacobian_encoded_i[5] * jacobian_encoded_j[jac_row];
			}			
		}


		{
			twist_gradient_i.rotation = make_float3(r.y, -r.x, 0.0f);
			twist_gradient_i.translation = make_float3(0.0f, 0.0f, 1.0f);
			twist_gradient_j.rotation = make_float3(-s.y, s.x, 0.0f);
			twist_gradient_j.translation = make_float3(0.0f,  0.0f, -1.0f);
#pragma unroll
			for (int jac_row = 0; jac_row < 6; jac_row++) {
				jtj_blk[6 * jac_row + 0] += weight_square * jacobian_encoded_i[0] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 1] += weight_square * jacobian_encoded_i[1] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 2] += weight_square * jacobian_encoded_i[2] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 3] += weight_square * jacobian_encoded_i[3] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 4] += weight_square * jacobian_encoded_i[4] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 5] += weight_square * jacobian_encoded_i[5] * jacobian_encoded_j[jac_row];
			}			
		}
	}


	__device__ __forceinline__ void computeChannelledJtJBlock(
		const float jacobian_channelled[18],
		float jtj_blk[jtj_blk_size],
		const float weight_square = 1.0f
	) {
		//The first iteration: assign
		const float* jacobian = jacobian_channelled;
#pragma unroll
		for (int jac_row = 0; jac_row < 6; jac_row++) {
			jtj_blk[6 * jac_row + 0] = weight_square * jacobian[0] * jacobian[jac_row];
			jtj_blk[6 * jac_row + 1] = weight_square * jacobian[1] * jacobian[jac_row];
			jtj_blk[6 * jac_row + 2] = weight_square * jacobian[2] * jacobian[jac_row];
			jtj_blk[6 * jac_row + 3] = weight_square * jacobian[3] * jacobian[jac_row];
			jtj_blk[6 * jac_row + 4] = weight_square * jacobian[4] * jacobian[jac_row];
			jtj_blk[6 * jac_row + 5] = weight_square * jacobian[5] * jacobian[jac_row];
		}
		

		//The next 2 iterations: plus
		for(auto channel = 1; channel < 3; channel++) {
			jacobian = &(jacobian_channelled[channel * 6]);
#pragma unroll
			for (int jac_row = 0; jac_row < 6; jac_row++) {
				jtj_blk[6 * jac_row + 0] += weight_square * jacobian[0] * jacobian[jac_row];
				jtj_blk[6 * jac_row + 1] += weight_square * jacobian[1] * jacobian[jac_row];
				jtj_blk[6 * jac_row + 2] += weight_square * jacobian[2] * jacobian[jac_row];
				jtj_blk[6 * jac_row + 3] += weight_square * jacobian[3] * jacobian[jac_row];
				jtj_blk[6 * jac_row + 4] += weight_square * jacobian[4] * jacobian[jac_row];
				jtj_blk[6 * jac_row + 5] += weight_square * jacobian[5] * jacobian[jac_row];
			}
		}
	}


	//The deprecated method
	__device__ __forceinline__ void computeSmoothJtJBlockOnline(
		const NodeGraphSmoothTerm2Jacobian& term2jacobian,
		unsigned typed_term,
		unsigned encoded_pair,
		float jtj_blk[jtj_blk_size],
		const float weight_square = 1.0f
	) {
		const ushort2 node_ij = term2jacobian.node_graph[typed_term];
		const auto xj4 = term2jacobian.reference_node_coords[node_ij.y];
		DualQuaternion dq_i = term2jacobian.node_se3[node_ij.x];
		DualQuaternion dq_j = term2jacobian.node_se3[node_ij.y];
		const mat34 Ti = dq_i.se3_matrix();
		const mat34 Tj = dq_j.se3_matrix();
		unsigned node_i, node_j;
		decode_nodepair(encoded_pair, node_i, node_j);

		//Explicit compute jacobian
		const float3 xj = make_float3(xj4.x, xj4.y, xj4.z);
		const float3 r = Ti.rot * xj + Ti.trans;
		const float3 s = Tj.rot * xj + Tj.trans;
		TwistGradientOfScalarCost twist_gradient_i, twist_gradient_j;
		
		//The order of two terms
		const float* jacobian_encoded_i;
		const float* jacobian_encoded_j;
		if(node_i == node_ij.x) {
			jacobian_encoded_i = (const float*)(&twist_gradient_i);
			jacobian_encoded_j = (const float*)(&twist_gradient_j);
		} else {
			jacobian_encoded_i = (const float*)(&twist_gradient_j);
			jacobian_encoded_j = (const float*)(&twist_gradient_i);
		}

		//The first iteration assign
		{
			twist_gradient_i.rotation = make_float3(0.0f, r.z, -r.y);
			twist_gradient_i.translation = make_float3(1.0f, 0.0f, 0.0f);
			twist_gradient_j.rotation = make_float3(0.0f, -s.z, s.y);
			twist_gradient_j.translation = make_float3(-1.0f,  0.0f,  0.0f);
#pragma unroll
			for (int jac_row = 0; jac_row < 6; jac_row++) {
				jtj_blk[6 * jac_row + 0] = weight_square * jacobian_encoded_i[0] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 1] = weight_square * jacobian_encoded_i[1] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 2] = weight_square * jacobian_encoded_i[2] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 3] = weight_square * jacobian_encoded_i[3] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 4] = weight_square * jacobian_encoded_i[4] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 5] = weight_square * jacobian_encoded_i[5] * jacobian_encoded_j[jac_row];
			}
		}

		//The next two iterations, plus
		{
			twist_gradient_i.rotation = make_float3(-r.z, 0.0f, r.x);
			twist_gradient_i.translation = make_float3(0.0f, 1.0f, 0.0f);
			twist_gradient_j.rotation = make_float3(s.z, 0.0f, -s.x);
			twist_gradient_j.translation = make_float3( 0.0f, -1.0f,  0.0f);
#pragma unroll
			for (int jac_row = 0; jac_row < 6; jac_row++) {
				jtj_blk[6 * jac_row + 0] += weight_square * jacobian_encoded_i[0] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 1] += weight_square * jacobian_encoded_i[1] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 2] += weight_square * jacobian_encoded_i[2] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 3] += weight_square * jacobian_encoded_i[3] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 4] += weight_square * jacobian_encoded_i[4] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 5] += weight_square * jacobian_encoded_i[5] * jacobian_encoded_j[jac_row];
			}			
		}


		{
			twist_gradient_i.rotation = make_float3(r.y, -r.x, 0.0f);
			twist_gradient_i.translation = make_float3(0.0f, 0.0f, 1.0f);
			twist_gradient_j.rotation = make_float3(-s.y, s.x, 0.0f);
			twist_gradient_j.translation = make_float3(0.0f,  0.0f, -1.0f);
#pragma unroll
			for (int jac_row = 0; jac_row < 6; jac_row++) {
				jtj_blk[6 * jac_row + 0] += weight_square * jacobian_encoded_i[0] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 1] += weight_square * jacobian_encoded_i[1] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 2] += weight_square * jacobian_encoded_i[2] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 3] += weight_square * jacobian_encoded_i[3] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 4] += weight_square * jacobian_encoded_i[4] * jacobian_encoded_j[jac_row];
				jtj_blk[6 * jac_row + 5] += weight_square * jacobian_encoded_i[5] * jacobian_encoded_j[jac_row];
			}			
		}
	}

	__global__ void computeJtJNonDiagonalBlockNoSyncKernel(
		const NodePair2TermsIndex::NodePair2TermMap nodepair2term,
		const Term2JacobianMaps term2jacobian,
		float* jtj_blks,
		const PenaltyConstants constants = PenaltyConstants()
	) {
		const auto nodepair_idx = blockIdx.x;
		const auto encoded_pair = nodepair2term.encoded_nodepair[nodepair_idx];
		const auto term_begin = nodepair2term.nodepair_term_range[nodepair_idx].x;
		const auto term_end = nodepair2term.nodepair_term_range[nodepair_idx].y;
		const auto term_size = term_end - term_begin;
		const auto padded_term_size = thread_blk_size * ((term_size + thread_blk_size - 1) / thread_blk_size);
		const auto warp_id = threadIdx.x >> 5;
		const auto lane_id = threadIdx.x & 31;

		//The shared memory for reduction
		__shared__ float shared_blks[jtj_blk_size][num_warps];

		//Zero out the elements
		for(auto iter = threadIdx.x; iter < jtj_blk_size * num_warps; iter += thread_blk_size) {
			shared_blks[iter % jtj_blk_size][iter / jtj_blk_size] = 0.0f;
		}
		__syncthreads();

		
		for (auto iter = threadIdx.x; iter < padded_term_size; iter += thread_blk_size)
		{
			//The global term index
			bool term_valid = true;
			//The memory for store the JtResidual result of each threads
			float local_blks[jtj_blk_size];
			
			if(iter < term_size)
			{
				const auto term_idx = nodepair2term.nodepair_term_index[term_begin + iter];
				unsigned typed_term_idx;
				TermType term_type;
				query_typed_index(term_idx, nodepair2term.term_offset, term_type, typed_term_idx);

				switch (term_type) {
				case TermType::DenseImage:
					{
						float term_jacobian[6] = {0};
						float nodepair_weight = 0;
						computeScalarJtJBlockJacobian(term2jacobian.dense_depth_term, encoded_pair, typed_term_idx, term_jacobian, &nodepair_weight);
						computeScalarJtJBlock(term_jacobian, local_blks, constants.DenseDepthSquared() * nodepair_weight);
					}
					break;
				case TermType::Smooth:
					computeSmoothJtJBlock(term2jacobian.smooth_term, typed_term_idx, encoded_pair, local_blks, constants.SmoothSquared());
					break;
				/*case TermType::DensityMap:
					{
						float term_jacobian[6] = {0};
						float nodepair_weight = 0;
						computeScalarJtJBlockJacobian(term2jacobian.density_map_term, encoded_pair, typed_term_idx, term_jacobian, &nodepair_weight);
						computeScalarJtJBlock(term_jacobian, local_blks, constants.DensitySquared() * nodepair_weight);
					}
					break;*/
				case TermType::Foreground:
					{
						float term_jacobian[6] = {0};
						float nodepair_weight = 0;
						computeScalarJtJBlockJacobian(term2jacobian.foreground_mask_term, encoded_pair, typed_term_idx, term_jacobian, &nodepair_weight);
						computeScalarJtJBlock(term_jacobian, local_blks, constants.ForegroundSquared() * nodepair_weight);
					}
					break;
				case TermType::Feature:
					{
						float term_jacobian[18] = {0};
						float nodepair_weight = 0;
						computeFeatureJtJBlockJacobian(term2jacobian.sparse_feature_term, encoded_pair, typed_term_idx, term_jacobian, &nodepair_weight);
						computeChannelledJtJBlock(term_jacobian, local_blks, constants.SparseFeatureSquared() * nodepair_weight);
					}
					break;
				default:
					term_valid = false;
					break;
				}
			}

			//__syncthreads();

			//Do a reduction
			for (int i = 0; i < jtj_blk_size; i++) {
				float data = (iter < term_size && term_valid) ? local_blks[i] : 0.0f;
				data = warp_scan(data);
				if (lane_id == warpSize - 1) {
					shared_blks[i][warp_id] += data;
				}
			}
		}

		__syncthreads();

		//Write to output
		for(auto iter = threadIdx.x; iter < jtj_blk_size; iter += thread_blk_size) 
			jtj_blks[jtj_blk_size * nodepair_idx + iter] = (shared_blks[iter][0] + shared_blks[iter][1] + shared_blks[iter][2] + shared_blks[iter][3]);
	}


} // namespace device
} // namespace surfelwarp

void surfelwarp::JtJMaterializer::computeNonDiagonalBlocksNoSync(hipStream_t stream)
{
	//Correct the size of node pairs
	const auto num_nodepairs = m_nodepair2term_map.encoded_nodepair.Size();
	SURFELWARP_CHECK_EQ(num_nodepairs, m_nodepair2term_map.nodepair_term_range.Size());
	m_nondiag_blks.ResizeArrayOrException(num_nodepairs * device::jtj_blk_size);
	
	//Invoke the kernel
	dim3 blk(device::thread_blk_size);
	dim3 grid(num_nodepairs);
	device::computeJtJNonDiagonalBlockNoSyncKernel<<<grid, blk, 0, stream>>>(
		m_nodepair2term_map,
		m_term2jacobian_map,
		m_nondiag_blks.Ptr(),
		m_penalty_constants
	);

	//Sync and check error
#if defined(CUDA_DEBUG_SYNC_CHECK)
	cudaSafeCall(hipStreamSynchronize(stream));
	cudaSafeCall(hipGetLastError());
#endif
	
	//Do a sanity check
	//nonDiagonalBlocksSanityCheck();
}